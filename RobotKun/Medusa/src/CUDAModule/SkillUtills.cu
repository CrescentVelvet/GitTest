#include <hip/hip_runtime.h>



#include <math.h>
#include <stdio.h>
#include <time.h>
#include <iostream>

#define FRAME_PERIOD (1 / 75.0)
#define ZERO_NUM (1e-8)
#define A_FACTOR (1.5)
#define PI (3.14159265359)
#define G (9.8)
#define SQRT_2 (1.414)
#define PLAYER_SIZE (90.0)

#define MAX_PLAYER (16)
#define THREAD_NUM_PASS (128)
#define BLOCK_X_PASS (16)
#define BLOCK_Y_PASS (MAX_PLAYER * 2)
#define MAX_BALL_SPEED (6500)
#define MIN_BALL_SPEED (1000)
#define BALL_SPEED_UNIT ((MAX_BALL_SPEED - MIN_BALL_SPEED) / BLOCK_X_PASS)

//#define MAX_CHIP_SPEED (400)
//#define MIN_CHIP_SPEED (50)
//#define CHIP_SPEED_UNIT ((MAX_CHIP_SPEED - MIN_CHIP_SPEED) / BLOCK_X_PASS)

#define MIN_DELTA_TIME (0)
#define OUR_RESPONSE_TIME (0.0)
#define THEIR_RESPONSE_TIME (0.0)
#define CAN_NOT_GET_STOP_BALL (true)

//---外部參數---
//场地参数
#define PITCH_LENGTH (9000.0)
#define PITCH_WIDTH (6000.0)
#define PENALTY_LENGTH (1000.0)
#define PENALTY_WIDTH (2000.0)
#define FIELD_BUFFER (300.0)
#define PENALTY_BUFFER (200.0)
//摩擦參數
#define ROLLING_FRACTION_REAL (800)
#define ROLLING_FRACTION_SIM (1520)
#define SLIDING_FRACTION_REAL (ROLLING_FRACTION_REAL * 15.0)
#define SLIDING_FRACTION_SIM (ROLLING_FRACTION_SIM * 15.0)
#define FLAT_ROLL_RATIO (5.0/7.0)
#define FLAT_SLIDE_RATIO (2.0/7.0)
//車球參數
#define PLAYER_CENTER_TO_BALL_CENTER (60)
#define STEP_TIME (0.2)
#define AVOID_DIST (2*PLAYER_SIZE)
//------------

//---CUDA參數---
#define PLAYER_NUM (16)
#define ANGLE_NUM (12)
#define DIST_NUM (8)
#define STEP_DIST (175)
#define SAFE_DIST (3.9*PLAYER_SIZE)
#define IGNORE_DIST (4*PLAYER_SIZE)
#define IGNORE_GOALIE (1)
#define IGNORE_GUARD (1)
//-------------

//时间预测的运动学参数
#define OUR_MAX_SPEED (3000)
#define OUR_MAX_ACC (4500)
#define OUR_MAX_DEC (4500)

#define THEIR_MAX_SPEED (3000)
#define THEIR_MAX_ACC (4500)
#define THEIR_MAX_DEC (4500)

#define OUR_PREDICT_BUFFER_TIME (0.0)
#define THEIR_PREDICT_BUFFER_TIME (0.5)

#define MAX_FLAT_VEL (5000.0)
#define MIN_FLAT_VEL (2000.0)
#define MAX_CHIP_VEL (4000.0)
#define MIN_CHIP_VEL (500.0)

#define True (1)
#define False (0)

//#define CHIP_ENERGY_LEFT_1 (0.34) //挑球后与地面碰撞后的能量剩余比例
//#define CHIP_ENERGY_LEFT_2 (0.97)

//计算点位评分
//#define BLOCK_X_FOR_POS_SCORE (4)
//#define BLOCK_Y_FOR_POS_SCORE (4)
//#define THREAD_X_FOR_POS_SCORE (32)
//#define THREAD_Y_FOR_POS_SCORE (32)
#define INITIAL_VALUE (99999)

// 评估函数各项的阈值
//__constant__ float maxDistToGoal =  900000;//sqrt(pow(PITCH_LENGTH, 2) + pow(PITCH_WIDTH, 2))
//__constant__ float minShootAngle = 0;
//__constant__ float maxShootAngle = PI / 2;
//__constant__ float maxDistToBall = 900000;//sqrt(pow(PITCH_LENGTH, 2) + pow(PITCH_WIDTH, 2))
//__constant__ float minDistToPassLine = 10.0;
//__constant__ float maxDistToPassLine = 900000;//sqrt(pow(PITCH_LENGTH, 2) + pow(PITCH_WIDTH, 2))
//__constant__ float minDistToEnemy = 30.0;
//__constant__ float maxDistToEnemy = 900000;//sqrt(pow(PITCH_LENGTH, 2) + pow(PITCH_WIDTH, 2))
// 评估函数各项的权重
//__constant__ float weight1 = 5;// 1.距离对方球门的距离
//__constant__ float weight2 = 8;// 2.射门有效角度
//__constant__ float weight3 = 0.5;// 3.跟球的距离
//__constant__ float weight4 = 0.3;// 4.对方车到传球线的距离
//__constant__ float weight5 = 0.5;// 5.对方车到接球点的距离
// 挑球模型参数
#define CHIP_FIRST_ANGLE (54.29 / 180.0 * PI)
#define CHIP_SECOND_ANGLE (45.59 / 180.0 * PI)
#define CHIP_VEL_RATIO  (0.6372)
#define MIN_CHIP_DIST (0.5)
#define MAX_CHIP_DIST (4.0)
#define MAX_CHIP_SPEED (50.0 * sqrt(2*G*MAX_CHIP_DIST/sin(2*CHIP_FIRST_ANGLE)))
#define MIN_CHIP_SPEED (50.0 * sqrt(2*G*MIN_CHIP_DIST/sin(2*CHIP_FIRST_ANGLE)))
#define CHIP_SPEED_UNIT ((MAX_CHIP_SPEED - MIN_CHIP_SPEED) / BLOCK_X_PASS)

#define CHIP_DIST_RATIO (0.8)
#define CHIP_LENGTH_RATIO (1.266)
typedef struct {
    float x, y;
} Vector;

typedef struct {
    float x, y;
} Point;

typedef struct {
    Point Pos;
    Vector Vel;
    bool isValid;
} Player;

typedef struct {
    Point interPos;
    bool isValid;
    float interTime;
    float Vel;
    float dir;
    int playerIndex;
    float deltaTime;
    float Q;
} nType;

typedef struct{

} DebugInfor;

typedef struct {
    Point p;
    float score;
} scoreAndPoint;


//計算垂點
__device__ Point projectionPointToLine(Point LP1, Point LP2, Point P) {
    Point result;
    if (LP2.x == LP1.x) {
        result.x = LP1.x;
        result.y = P.y;
    } else {
        // 如果该线段不平行于X轴也不平行于Y轴，则斜率存在且不为0。设线段的两端点为pt1和pt2，斜率为：
        float k = (LP2.y - LP1.y) / (LP2.x - LP1.x);
        // 该直线方程为:					y = k* ( x - pt1.x) + pt1.y
        // 其垂线的斜率为 -1/k,垂线方程为:	y = (-1/k) * (x - point.x) + point.y
        // 联立两直线方程解得:
        result.x = (k * k * LP1.x + k * (P.y - LP1.y) + P.x) / (k * k + 1);
        result.y = k * (result.x - LP1.x) + LP1.y;
    }
    return result;
}
//---通用函數---
__device__ bool if_finite(float a) {
    return fabs(a) < INITIAL_VALUE;
}

__device__ bool IsInField(Point p, float buffer=500) {
    return (p.x > -PITCH_LENGTH / 2 + buffer && p.x < PITCH_LENGTH / 2 - buffer
            && p.y < PITCH_WIDTH / 2 - buffer && p.y > -PITCH_WIDTH / 2 + buffer);
}

__device__ bool IsInPenalty(Point p, float buffer=300) {
    return (p.x < -PITCH_LENGTH/2 + PENALTY_LENGTH + buffer && p.x > -PITCH_LENGTH/2 &&
            p.y > -PENALTY_WIDTH/2 - buffer && p.y < PENALTY_WIDTH/2 + buffer) ||
           (p.x > PITCH_LENGTH/2 - PENALTY_LENGTH - buffer && p.x < PITCH_LENGTH/2 &&
            p.y > -PENALTY_WIDTH/2 - buffer && p.y < PENALTY_WIDTH/2 + buffer);
}
__device__ bool IsInEnemy(Point p1, Point p2, float buffer) {
    return ((p1.x-p2.x)*(p1.x-p2.x)+(p1.y-p2.y)*(p1.y-p2.y)) < (buffer*buffer);
}
__device__ bool IsOutOfLimit(const float value, const float upLimit, const float lowLimit, const float range){
    return (value > upLimit*(1-range) || value < lowLimit*(1+range));
}
__device__ float Dist(Point p1,Point p2){
    return sqrt((p1.x - p2.x) * (p1.x - p2.x) + (p1.y - p2.y) * (p1.y - p2.y));
}
//---球模型---
__device__ float flatRollTime(const float startVel, const float rollAcc){
    if(startVel < 0.0)
        return 0.0;
    return (startVel * FLAT_ROLL_RATIO / rollAcc);
}
__device__ float flatSlideTime(const float startVel, const float slideAcc){
    if(startVel < 0.0)
        return 0.0;
    return (startVel * FLAT_SLIDE_RATIO / slideAcc);
}
__device__ float flatRollDist(const float startVel, const float rollAcc){
    if (startVel < 0.0)
        return 0.0;
    return (pow(startVel * FLAT_ROLL_RATIO, 2.0) / (2.0 * rollAcc));
}
__device__ float flatSlideDist(const float startVel, const float slideAcc){
    if (startVel < 0.0)
        return 0.0;
    return ((pow(startVel, 2.0)-pow(startVel * FLAT_ROLL_RATIO, 2.0)) / (2.0 * slideAcc));
}
__device__ float flatStopTime(const float startVel, const float slideAcc, const float rollAcc){
    return flatSlideTime(startVel, slideAcc) + flatRollTime(startVel, rollAcc);
}
__device__ float flatMoveVel(const float startVel, const float time, const float slideAcc, const float rollAcc){
    const float slidingTime = flatSlideTime(startVel, slideAcc);
    const float stopTime = flatStopTime(startVel, slideAcc, rollAcc);
    float ballVel = 0;
    if(time > stopTime)
        ballVel = 0;
    else if (time < slidingTime)
        ballVel = startVel - slideAcc * time;
    else
        ballVel = startVel * FLAT_ROLL_RATIO - rollAcc * (time - slidingTime);
    return ballVel;
}

__device__ float flatMoveDist(const float startVel, const float time, const float slideAcc, const float rollAcc){
    float moveDist = 0;
    float slidingTime = flatSlideTime(startVel, slideAcc);
    float totalTime = flatStopTime(startVel, slideAcc, rollAcc);
    if(time > totalTime)
        moveDist = flatRollDist(startVel, rollAcc) + flatSlideDist(startVel, slideAcc);
    else if(time < slidingTime)
        moveDist = startVel * time - 0.5 * slideAcc * pow(time, 2.0);
    else {
        float slideDist = (pow(startVel, 2.0) - pow(startVel * FLAT_ROLL_RATIO, 2.0)) / (2.0 * slideAcc);
        float rollStartVel = startVel * FLAT_ROLL_RATIO;
        float rollTime = time - slidingTime;
        float rollDist = rollStartVel * rollTime - 0.5 * rollAcc * pow(rollTime, 2.0);
        moveDist = slideDist + rollDist;
    }
    return moveDist;
}
////////////////////////////////////////////////////////////////
/// \brief calculatFlatVel  計算球花給定時間走一段距離所需要的初速度
/// \param distance         距離
/// \param interTime        給定時間
/// \param isSim            是否為仿真
/// \return                 所需初速度
////////////////////////////////////////////////////////////////
__device__ float calculateFlatVel(float distance, float interTime, bool isSim)
{
    //interTime = max(interTime, 0.4);
    float ballSlidAcc = (isSim ? SLIDING_FRACTION_SIM : SLIDING_FRACTION_REAL) / 2;
    float ballRollAcc = (isSim ? ROLLING_FRACTION_SIM : ROLLING_FRACTION_REAL) / 2;
    const float minPassVel = sqrt(ballRollAcc * distance);
    float passVel = max((distance + 1.0/2.0*ballRollAcc*interTime*interTime) / interTime, minPassVel);
    passVel /= FLAT_ROLL_RATIO;
    //passVel = max(min(passVel, MAX_FLAT_VEL), MIN_FLAT_VEL);
    return passVel;
}
__device__ float calculateChipVel(float distance, float interTime, bool isSim)
{
    float passVel = 0;
    float chipDist = distance * CHIP_DIST_RATIO;
    passVel = chipDist / CHIP_LENGTH_RATIO;
    passVel = passVel > MAX_CHIP_VEL ? MAX_CHIP_VEL : passVel;
    passVel = passVel < MIN_CHIP_VEL ? MIN_CHIP_VEL : passVel;
    return passVel;
}
//------------


//////////////////////////////////////////////////////////////
/// \brief CUDA_compute_motion_1d   計算一維空間給定速度到點時間
/// \param x0                       目標距離
/// \param v0                       初速度
/// \param v1                       末速度
/// \param a_max                    最大加速度
/// \param d_max                    最大減速度
/// \param v_max                    最大速度
/// \param a_factor                 加速度因子
/// \param traj_time                到點時間
//////////////////////////////////////////////////////////////
__device__ void CUDA_compute_motion_1d(float x0, float v0, float v1, float a_max, float d_max,
                                       float v_max, float a_factor, float &traj_time)
{
    float traj_time_acc, traj_time_dec, traj_time_flat;
    traj_time_acc = traj_time_dec = traj_time_flat = 0.0;
    if ((x0 == 0 && v0 == v1) || !if_finite(x0) || !if_finite(v0) || !if_finite(v1)) {
        traj_time = 0;
        return;
    }

    a_max /= a_factor;
    d_max /= a_factor;

    float accel_dist_to_v1 = fabs((v1 + v0) / 2.0) * fabs(v1 - v0) / a_max;
    float decel_dist_to_v1 = fabs((v0 + v1) / 2.0) * fabs(v0 - v1) / d_max;

    float period = 1 / 40.0;

    if (v0 * x0 > 0 || (fabs(v0) > fabs(v1) && decel_dist_to_v1 > fabs(x0))) {
        float time_to_stop = fabs(v0) / (d_max);
        float x_to_stop = v0 * v0 / (2.0 * d_max);

        CUDA_compute_motion_1d(x0 + copysign(x_to_stop, v0), 0, v1, a_max * a_factor, d_max * a_factor, v_max, a_factor, traj_time);
        traj_time += time_to_stop;
        traj_time /= 1.25;
        return;
    }

    if (fabs(v0) > fabs(v1)) {
        traj_time_acc = (sqrt((d_max * v0 * v0 + a_max * (v1 * v1 + 2 * d_max * fabs(x0))) / (a_max + d_max)) - fabs(v0)) / a_max;

        if (traj_time_acc < 0.0)
            traj_time_acc = 0;
        traj_time_dec = ((fabs(v0) - fabs(v1)) + a_max * traj_time_acc) / d_max;
    }

    else if (accel_dist_to_v1 > fabs(x0)) {
        traj_time_acc = (sqrt(v0 * v0 + 2 * a_max * fabs(x0)) - fabs(v0)) / a_max;
        traj_time_dec = 0.0;
    }

    else {
        traj_time_acc = (sqrt((d_max * v0 * v0 + a_max * (v1 * v1 + 2 * d_max * fabs(x0))) / (a_max + d_max)) - fabs(v0)) / a_max;
        if (traj_time_acc < 0.0)
            traj_time_acc = 0;
        traj_time_dec = ((fabs(v0) - fabs(v1)) + a_max * traj_time_acc) / d_max;
    }


    if (traj_time_acc * a_max + fabs(v0) > v_max) {
        float dist_without_flat = (v_max * v_max - v0 * v0) / (2 * a_max) + (v_max * v_max - v1 * v1) / (2 * d_max);
        traj_time_flat = (fabs(x0) - dist_without_flat) / v_max;
    }
    else {
        traj_time_flat = 0;
    }

    if (FRAME_PERIOD * a_max + fabs(v0) > v_max && traj_time_flat > period) {
        traj_time = traj_time_flat + traj_time_dec;
    }
    else if (traj_time_acc < period && traj_time_dec == 0.0) {
        traj_time = traj_time_acc;
    }
    else if (traj_time_acc < period && traj_time_dec > 0.0) {
        traj_time = traj_time_dec;
    }
    else {
        traj_time = traj_time_acc + traj_time_flat / 1.1 + traj_time_dec / 1.1;
    }
}

//////////////////////////////////////////////////////////////
/// \brief CUDA_predictedTime   計算二維空間給定初始速度零速到點時間
/// \param x0                   出發點X
/// \param y0                   出發點Y
/// \param x1                   目標點X
/// \param y1                   目標點Y
/// \param vx                   初始速度X
/// \param vy                   初始速度Y
/// \param isTheir              是否為己方車
/// \return                     到點時間
//////////////////////////////////////////////////////////////
__device__ float CUDA_predictedTime(float x0, float y0, float x1, float y1, float vx, float vy, bool isTheir) {
    float timeX = 0.0;
    float timeY = 0.0;

    float Angle = atan2(vy+1e-5, vx+1e-5) - atan2(y0 - y1+1e-5, x0 - x1+1e-5);
//    float Angle = atan2(vy, vx) - atan2(y0 - y1, x0 - x1);
    float Speed = sqrt(vx * vx + vy * vy);
    vx = Speed * cospi(Angle / PI); //沿着球线
    vy = Speed * sinpi(Angle / PI); //切着球线

    float maxAcc, maxDec, maxSpeed;
    if(isTheir) {
        maxAcc = THEIR_MAX_ACC;
        maxDec = THEIR_MAX_DEC;
        maxSpeed = THEIR_MAX_SPEED;
    } else {
        maxAcc = OUR_MAX_ACC;
        maxDec = OUR_MAX_DEC;
        maxSpeed = OUR_MAX_SPEED;
    }
    CUDA_compute_motion_1d(sqrt((x1-x0)*(x1-x0)+(y1-y0)*(y1-y0)), vx, 0.0, maxAcc, maxDec, maxSpeed, 1.5, timeX);
    CUDA_compute_motion_1d(0, vy, 0.0, maxAcc, maxDec, maxSpeed, 1.5, timeY);
    if (timeX < 1e-5 || timeX > 50) timeX = 0;
    if (timeY < 1e-5 || timeY > 50) timeY = 0;
    //printf("(x0,y0):(%f,%f) (x1,y1):(%f,%f) (vx,vy):(%f,%f) (tX,tY):(%f,%f)\n",x0,y0,x1,y1,vx,vy,timeX,timeY);
    return (timeX > timeY ? timeX : timeY);
}

__device__ bool CUDA_predictedFlatInterTime(Point mePoint, Point ballPoint, Vector meVel, Vector ballVel,
                                            const float ballArriveTime, Point* interceptPoint, float* interTime,
                                            float responseTime, bool isTheir, bool isSim) {
    //meVel.x=0;
    //meVel.y=0;
    float ballRollAcc = (isSim ? ROLLING_FRACTION_SIM : ROLLING_FRACTION_REAL) / 2;
    float ballSlideAcc = (isSim ? SLIDING_FRACTION_SIM : SLIDING_FRACTION_REAL) / 2;
    const float originVel = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
    const float maxMoveTime = flatStopTime(originVel, ballSlideAcc, ballRollAcc);
    float testMoveDist = 0;
    float testMoveTime = 0;
    Point testPoint = ballPoint;
    bool canInter = false;
    bool safeFlag = true;
    float adjustLength = 0, interT=INITIAL_VALUE;
    for (testMoveTime = 0; testMoveTime < maxMoveTime; testMoveTime += STEP_TIME ) {
        testMoveDist = flatMoveDist(originVel, testMoveTime, ballSlideAcc, ballRollAcc);
        // 计算截球点
        testPoint.x = ballPoint.x + testMoveDist * ballVel.x / originVel;
        testPoint.y = ballPoint.y + testMoveDist * ballVel.y / originVel;
        // 对敌方截球点进行特殊处理
//        if(isTheir) {
//            Vector adjustDir;
//            adjustDir.x = mePoint.x - testPoint.x;
//            adjustDir.y = mePoint.y - testPoint.y;
//            adjustLength = sqrt(adjustDir.x * adjustDir.x + adjustDir.y * adjustDir.y);
//            testPoint.x += adjustDir.x / adjustLength * (AVOID_DIST>adjustLength?adjustLength:AVOID_DIST);
//            testPoint.y += adjustDir.y / adjustLength * (AVOID_DIST>adjustLength?adjustLength:AVOID_DIST);
//        }
        if(IsInPenalty(testPoint, PENALTY_BUFFER))
            continue;
        if (!IsInField(testPoint, FIELD_BUFFER)){
            canInter = false;
            break;
        }
        if(testMoveTime >= STEP_TIME * 3 && isTheir && IsInEnemy(testPoint,mePoint,SAFE_DIST)){
            safeFlag = false;
            break;
        }
        // 计算截球时间
        interT = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y, isTheir);
        if(interT + responseTime < testMoveTime){
            canInter = true;
            break;
        }
    }

    if(!safeFlag){
        interceptPoint->x = INITIAL_VALUE;
        interceptPoint->y = INITIAL_VALUE;
        *interTime = 0;
        return true;
    }
    // 无法截球
    if(!canInter) {
        interceptPoint->x = INITIAL_VALUE;
        interceptPoint->y = INITIAL_VALUE;
        *interTime = INITIAL_VALUE;
        return false;
    }
    // 能够截球计算截球时间和截球点
    *interceptPoint = testPoint;
    *interTime = interT;
    return true;
}

__device__ bool CUDA_FlatSecurityCheck(Point mePoint, Point ballPoint, Point target, Vector meVel, Vector ballVel,
                                       int enemyRole, float responseTime,
                                       float ignoreCloseEnemyDist, bool ignoreTheirGoalie, bool ignoreTheirGuard,
                                       bool isSim){
    Point passLine;
    float ballSpeed = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
    passLine.x = target.x - ballPoint.x;
    passLine.y = target.y - ballPoint.y;
    float passLineDist = sqrt(passLine.x * passLine.x + passLine.y * passLine.y);
    float passLineDir = atan2(passLine.y+1e-5, passLine.x+1e-5);
    float ballRollAcc = (isSim ? ROLLING_FRACTION_SIM : ROLLING_FRACTION_REAL) / 2;
    float ballSlideAcc = (isSim ? SLIDING_FRACTION_SIM : SLIDING_FRACTION_REAL) / 2;
    if(Dist(mePoint, ballPoint) < ignoreCloseEnemyDist && !IsInPenalty(mePoint, 0))
        return true;
    Point pedal = projectionPointToLine(ballPoint, target, mePoint);
    if(pedal.x > min(ballPoint.x,target.x) && pedal.x < max(ballPoint.x,target.x)){
        if(Dist(mePoint, pedal) < SAFE_DIST) return false;
    }
    else{
        if(min(Dist(mePoint,ballPoint),Dist(mePoint,target)) < SAFE_DIST) return false;
    }
    if(enemyRole == 1 && ignoreTheirGoalie) return true;
    if(enemyRole == 2 && ignoreTheirGuard) return true;
    float ballMoveTime = 0;
    while(1){
        ballMoveTime += STEP_TIME;
        float ballMoveDist = flatMoveDist(ballSpeed, ballMoveTime, ballSlideAcc, ballRollAcc);
        Point ballPos;
        ballPos.x = ballPoint.x + ballMoveDist * cos(passLineDir);
        ballPos.y = ballPoint.y + ballMoveDist * sin(passLineDir);
        if(!IsInField(ballPos) || ballMoveDist > passLineDist)
            return true;
        float adjustDir = atan2(mePoint.y - ballPos.y+1e-5, mePoint.x - ballPos.x+1e-5);
        Point testPoint;
        testPoint.x = ballPos.x + SAFE_DIST * cos(adjustDir);
        testPoint.y = ballPos.y + SAFE_DIST * sin(adjustDir);
        float enemyTime = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y, True);
        if(enemyTime + responseTime - ballMoveTime < 0)
            return false;
    }
    return true;
}

__device__ bool CUDA_predictedChipInterTime(Point mePoint, Point ballPoint, Vector meVel, Vector ballVel,
                                            const float ballArriveTime, Point* interceptPoint, float* interTime,
                                            float responseTime, bool isTheir, bool isSim) {
    float chipVel = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y);
    float meArriveTime = INITIAL_VALUE;
    float ballAcc = (isSim ? ROLLING_FRACTION_SIM : ROLLING_FRACTION_REAL) / 2.0;
    float ballAccSecondJump = 0;
    float stepTime = 0.05;
    float testBallLength = 0;
    Point testPoint = ballPoint;

    // 挑球第一段的时间, 单位s
    float time_1 = 2.0 * chipVel * sin(CHIP_FIRST_ANGLE) / 1000.0 / G;
    // 挑球第一段的距离, 单位m
    float length_1 = 1.0 / 2 * G * time_1 * time_1 / tan(CHIP_FIRST_ANGLE);
    // 挑球第二段的距离, 单位m
    float length_2 = (CHIP_LENGTH_RATIO - 1.0) * length_1;
    // 挑球第二段的时间, 单位s
    float time_2 = sqrt(2 * length_2 * tan(CHIP_SECOND_ANGLE) / G); // 单位s
    // 挑球第一二段的距离, 单位mm
    length_1 *= 1000;
    length_2 *= 1000;
    // 可以开始截球的起始距离
    float jumpDist = 0;
    // 可以开始截球的起始速度
    float moveVel = 0;
    // 可以开始截球的起始时刻
    float ballDropTime = 0;
    // 球滚动的最大时间
    float max_time = 0;
    if(isTheir) {
        jumpDist = length_1;
        moveVel = length_2 / time_2;
        ballDropTime = time_1;
        max_time = time_2 + chipVel * chipVel * CHIP_VEL_RATIO / 980 / ballAcc;
    } else {
        jumpDist = length_1 + length_2;
        moveVel = chipVel * chipVel * CHIP_VEL_RATIO / 980;
        ballDropTime = time_1 + time_2;
        max_time = moveVel / ballAcc;
    }

    bool canInter = true, theirCanTouch = false, isSecondJump = true;
    float afterArrivedTime = 0, secondJumpDist = 0, secondJumpTime = 0, secondJumpVelLeft = 0;
    while (afterArrivedTime < max_time) {
        Vector direc;
        if(isTheir) {
            if(isSecondJump) {
                testBallLength = jumpDist + moveVel * afterArrivedTime;
                if(testBallLength > length_1 + length_2) {
                    secondJumpDist = length_2;
                    secondJumpTime = time_2;
                    secondJumpVelLeft = chipVel * chipVel * CHIP_VEL_RATIO / 980;
                    isSecondJump = false;
                }
            }
            else {
                testBallLength = jumpDist + secondJumpDist + (secondJumpVelLeft * (afterArrivedTime - secondJumpTime) - 0.5 * ballAcc * (afterArrivedTime - secondJumpTime) * (afterArrivedTime - secondJumpTime));
            }
        }
        else {
            testBallLength = jumpDist + (moveVel * afterArrivedTime - 0.5 * ballAcc * afterArrivedTime * afterArrivedTime);
        }

        direc.x = testBallLength * ballVel.x / chipVel;
        direc.y = testBallLength * ballVel.y / chipVel;
        testPoint.x = ballPoint.x + direc.x;
        testPoint.y = ballPoint.y + direc.y;

        if(isTheir) {
            if(sqrt((mePoint.x - testPoint.x) * (mePoint.x - testPoint.x) + (mePoint.y - testPoint.y) * (mePoint.y - testPoint.y)) < PLAYER_SIZE * 1.2) {
                theirCanTouch = true;
                break;
            } else {
                Vector adjustDir;
                adjustDir.x = mePoint.x - testPoint.x;
                adjustDir.y = mePoint.y - testPoint.y;
                float length = sqrt(adjustDir.x * adjustDir.x + adjustDir.y * adjustDir.y);
                adjustDir.x /= length;
                adjustDir.y /= length;
                testPoint.x += adjustDir.x * PLAYER_SIZE;
                testPoint.y += adjustDir.y * PLAYER_SIZE;
            }
        }

        meArriveTime = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y, isTheir);

        if(meArriveTime < 0.10) meArriveTime = 0;

        if(IsInPenalty(testPoint, 200)) {
            afterArrivedTime += stepTime;
            continue;
        }
        if (!IsInField(testPoint)) {
            canInter = false;
            break;
        }
        if(meArriveTime + responseTime < ballDropTime + afterArrivedTime) {
            break;
        }
        afterArrivedTime += stepTime;
    }

    if(!canInter || (CAN_NOT_GET_STOP_BALL && afterArrivedTime >= max_time)){
        interceptPoint->x = INITIAL_VALUE;
        interceptPoint->y = INITIAL_VALUE;
        *interTime = INITIAL_VALUE;
        return false;
    }
    *interceptPoint = testPoint;
    *interTime = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y, isTheir);
    *interTime = *interTime > ballDropTime ? *interTime : ballDropTime;
//    Vector velDir;
//    velDir.x = ballVel.x / chipVel;
//    velDir.y = ballVel.y / chipVel;
////    float meX = 547;
////    float meY = -276;
//    float meX = 444;
//    float meY = -105;

//    float vel = 428;
//    float dir = 4.07;
//    if(!isTheir && mePoint.x < meX + 5 && mePoint.x > meX - 5 && mePoint.y < meY + 5 && mePoint.y > meY - 5 && chipVel > vel - 1.0 && chipVel < vel + 1.0  && atan(ballVel.y / ballVel.x) > dir - 0.1 -  PI && atan(ballVel.y / ballVel.x) < dir + 0.1 - PI) {
//        printf("%lf %lf (%lf, %lf), (%lf, %lf), %lf, %lf, interTime: %f\n", testPoint.x, testPoint.y, ballPoint.x + velDir.x * length_1, ballPoint.y + velDir.y * length_1, ballPoint.x + velDir.x * (length_2 + length_1), ballPoint.y + velDir.y * (length_2 + length_1), chipVel, atan(ballVel.y / ballVel.x), *interTime);
//    }


    if(theirCanTouch){
        *interTime = 0.0;
    }

//    float vel = 202;
//    float dir = 3.28885;
//    if(isTheir && ballPoint.x + velDir.x * length_1 < 0 && chipVel > vel - 1.0 && chipVel < vel + 1.0  && atan(ballVel.y / ballVel.x) > dir - 0.001 - PI && atan(ballVel.y / ballVel.x) < dir + 0.001 - PI) {
//        printf("%lf, %lf, (%lf, %lf)\n", *interTime, testBallLength, (*interceptPoint).x, (*interceptPoint).y);
//    }

    return true;
}

__device__ bool CUDA_ChipSecurityCheck(Point mePoint, Point ballPoint, Point target, Vector meVel, Vector ballVel,
                                        int enemyRole, float responseTime,
                                        bool ignoreTheirGuard, bool isSim){
    float BALL_ACC = (isSim ? ROLLING_FRACTION_SIM : ROLLING_FRACTION_REAL) / 2.0;

    Point passLine;
    passLine.x = target.x - ballPoint.x;
    passLine.y = target.y - ballPoint.y;
    float passLineDist = sqrt(passLine.x * passLine.x + passLine.y * passLine.y) - PLAYER_SIZE;
    float passLineDir = atan2(passLine.y+1e-5, passLine.x+1e-5);

    float chipLength1 = sqrt(ballVel.x * ballVel.x + ballVel.y * ballVel.y) / 1000;
    float chipTime1 = sqrt(2.0 * chipLength1 * tan(CHIP_FIRST_ANGLE) / G);
    float chipLength2 = (CHIP_LENGTH_RATIO - 1) * chipLength1;
    float chipTime2 = sqrt(2 * chipLength2 * tan(CHIP_SECOND_ANGLE) / G);
    chipLength1 *= 1000;
    chipLength2 *= 1000;
    Point startPos;
    startPos.x = ballPoint.x + (chipLength1 + chipLength2) * cos(passLineDir);
    startPos.y = ballPoint.y + (chipLength1 + chipLength2) * sin(passLineDir);
    float startTime = chipTime1 + chipTime2;
    float startVel = pow(chipTime1 * 1000 * G / (2 * sin(CHIP_FIRST_ANGLE)), 2) * CHIP_VEL_RATIO / 9800;
    float startDist = Dist(startPos, ballPoint);
    Point pedal = projectionPointToLine(ballPoint, target, mePoint);
    if(pedal.x > min(startPos.x,target.x) && pedal.x < max(startPos.x,target.x)){
        if(Dist(mePoint, pedal) < SAFE_DIST) return false;
    }
    else{
        if(min(Dist(mePoint,startPos),Dist(mePoint,target)) < SAFE_DIST) return false;
    }
    if(enemyRole == 2 && ignoreTheirGuard) return true;
    Point ballPos = startPos;
    float ballSpeed = startVel;
    float ballMoveTime = startTime;
    float ballMoveDist = startDist;
    while(1){
        ballMoveTime += STEP_TIME;
        ballSpeed = startVel - BALL_ACC * (ballMoveTime - startTime);
        if(ballSpeed < 0) ballSpeed = 0;
        ballMoveDist = startDist + (ballSpeed + startVel) / 2 * (ballMoveTime - startTime);
        ballPos.x = ballPoint.x + ballMoveDist * cos(passLineDir);
        ballPos.y = ballPoint.y + ballMoveDist * sin(passLineDir);
        if(!IsInField(ballPos) || ballMoveDist > passLineDist)
            return true;
        float adjustDir = atan2(mePoint.y - ballPos.y+1e-5, mePoint.x - ballPos.x+1e-5);
        Point testPoint;
        testPoint.x = ballPos.x + SAFE_DIST * cos(adjustDir);
        testPoint.y = ballPos.y + SAFE_DIST * sin(adjustDir);
        float enemyTime = CUDA_predictedTime(mePoint.x, mePoint.y, testPoint.x, testPoint.y, meVel.x, meVel.y, True);
        if(enemyTime + responseTime - ballMoveTime < 0)
            return false;
    }
    return true;
}

__global__ void calculateAllInterInfoV10(Player* players, Point* ballPos, nType* bestPass, int leader, double theirFlatResponseTime, double theirChipResponseTime, bool isSim) {

    //printf("bushibawokao");

    int catcherIndex = blockIdx.x;
    int angleIndex = blockIdx.y;
    int distanceIndex =  threadIdx.x / PLAYER_NUM;
    int blockerIndex = threadIdx.x % PLAYER_NUM;
    bool flatSecurity = true;
    bool chipSecurity = true;
    bool canInter;
    float interTime = 0.0;
    Point interPoint;

    Point catchPoint;
    catchPoint.x = players[catcherIndex].Pos.x + (distanceIndex + 2) * STEP_DIST * cospi(2.0 * angleIndex / ANGLE_NUM);
    catchPoint.y = players[catcherIndex].Pos.y + (distanceIndex + 2) * STEP_DIST * sinpi(2.0 * angleIndex / ANGLE_NUM);

    if(!IsInField(catchPoint) || IsInPenalty(catchPoint) || !players[catcherIndex].isValid){
        flatSecurity = chipSecurity = false;
    }

    double selfPassBuffer = 1500.0;
    if(catcherIndex == leader && IsInPenalty(catchPoint, selfPassBuffer) && !IsInField(catchPoint, selfPassBuffer)) flatSecurity = chipSecurity = false;

    if(catcherIndex == leader && distanceIndex != 2) flatSecurity = chipSecurity = false;
    float catchtime = CUDA_predictedTime(players[catcherIndex].Pos.x, players[catcherIndex].Pos.y, catchPoint.x, catchPoint.y, players[catcherIndex].Vel.x, players[catcherIndex].Vel.y, False);
    float ballLineDist = sqrt((catchPoint.x - ballPos->x) * (catchPoint.x - ballPos->x) + (catchPoint.y - ballPos->y) * (catchPoint.y - ballPos->y));
    float ballLineAngle = atan2(catchPoint.y - ballPos->y, catchPoint.x - ballPos->x);
    Vector ballFlatVel, ballChipVel;
    float ballFlatSpeed = calculateFlatVel(ballLineDist, catchtime + OUR_PREDICT_BUFFER_TIME, isSim);
//    if (catcherIndex == leader) ballFlatSpeed = MIN_FLAT_VEL;
//    if(leader != catcherIndex) ballFlatSpeed = calculateFlatVel(ballLineDist, catchtime + OUR_PREDICT_BUFFER_TIME, isSim);
//    else ballFlatSpeed = calculateFlatVel(ballLineDist, catchtime + OUR_PREDICT_BUFFER_TIME + 0.5, isSim);
//    printf("minspeed:%f\n", ballFlatSpeed);
    if(players[catcherIndex].isValid && IsOutOfLimit(ballFlatSpeed, MAX_FLAT_VEL, MIN_FLAT_VEL, 0.05)){
        //flatSecurity = false;
        if(ballFlatSpeed > MAX_FLAT_VEL){
            ballFlatSpeed = MAX_FLAT_VEL;
            ballFlatVel.x = ballFlatSpeed * cos(ballLineAngle);
            ballFlatVel.y = ballFlatSpeed * sin(ballLineAngle);
            CUDA_predictedFlatInterTime(players[catcherIndex].Pos,*ballPos,players[catcherIndex].Vel,ballFlatVel,
                                        INITIAL_VALUE, &catchPoint,&catchtime,OUR_RESPONSE_TIME,false,isSim);
        }
    }
//    if (catcherIndex == leader) ballFlatSpeed = 1000.0;
    float ballChipSpeed = calculateChipVel(ballLineDist, catchtime + OUR_PREDICT_BUFFER_TIME, isSim);
    if(IsOutOfLimit(ballChipSpeed, MAX_CHIP_VEL, MIN_CHIP_VEL, 0.05)){
        chipSecurity = false;
    }
    ballFlatVel.x = ballFlatSpeed * cos(ballLineAngle);
    ballFlatVel.y = ballFlatSpeed * sin(ballLineAngle);
    ballChipVel.x = ballChipSpeed * cos(ballLineAngle);
    ballChipVel.y = ballChipSpeed * sin(ballLineAngle);

    int enemyRole = 0;
    if(IsInPenalty(players[blockerIndex + PLAYER_NUM].Pos, 0))
        enemyRole = 1;
    else
        if(IsInPenalty(players[blockerIndex + PLAYER_NUM].Pos, 300))
            enemyRole = 2;
    if(players[blockerIndex + PLAYER_NUM].isValid == 1 && flatSecurity){
          flatSecurity = CUDA_FlatSecurityCheck(players[blockerIndex + PLAYER_NUM].Pos,*ballPos,catchPoint,players[blockerIndex + PLAYER_NUM].Vel,ballFlatVel,
                         enemyRole,theirFlatResponseTime,IGNORE_DIST,IGNORE_GOALIE,IGNORE_GUARD,isSim);
    }

    if(players[blockerIndex + PLAYER_NUM].isValid == 1 && chipSecurity){
          chipSecurity = CUDA_ChipSecurityCheck(players[blockerIndex + PLAYER_NUM].Pos,*ballPos,catchPoint,players[blockerIndex + PLAYER_NUM].Vel,ballChipVel,
                         enemyRole,theirChipResponseTime,IGNORE_GUARD,isSim);
    }

    int offset = catcherIndex * ANGLE_NUM * DIST_NUM * PLAYER_NUM + angleIndex * DIST_NUM * PLAYER_NUM + distanceIndex * PLAYER_NUM + blockerIndex;

    bestPass[offset].interPos = catchPoint;
    bestPass[offset].interTime = catchtime;
    bestPass[offset].playerIndex = catcherIndex;
    bestPass[offset].dir = ballLineAngle;
    bestPass[offset].Vel = flatSecurity? ballFlatSpeed: 0;
    bestPass[offset].isValid = flatSecurity;

    offset += PLAYER_NUM * ANGLE_NUM * DIST_NUM * PLAYER_NUM;

    bestPass[offset].interPos = catchPoint;
    bestPass[offset].interTime = catchtime;
    bestPass[offset].playerIndex = catcherIndex;
    bestPass[offset].dir = ballLineAngle;
    bestPass[offset].Vel = chipSecurity? ballChipSpeed: 0;
    bestPass[offset].isValid = chipSecurity;
}

extern "C" void BestPass(Player* players, Point* ball, nType* result, int leader, double theirFlatResponseTime, double theirChipResponseTime, bool isSim) {
    nType *pointInfor;
    hipMallocManaged((void**)&pointInfor, 2*PLAYER_NUM*ANGLE_NUM*DIST_NUM*PLAYER_NUM*sizeof(nType));
//    cudaEvent_t start, stop;
//    cudaEventCreate(&start);
//    cudaEventCreate(&stop);
//    cudaEventRecord(start);

    //new CUDA
    //printf("angleNum%d,playerNum%d,distanceNum%d\n",params->angleNum,params->playerNum,params->distanceNum);

    dim3 blocks(PLAYER_NUM, ANGLE_NUM);
    calculateAllInterInfoV10 <<<blocks, DIST_NUM * PLAYER_NUM>>> (players,ball,pointInfor, leader, theirFlatResponseTime, theirChipResponseTime,isSim);
    hipDeviceSynchronize();
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess){
        printf("CUDA ERROR: %d\n", (int)cudaStatus);
        printf("Error Name: %s\n", hipGetErrorName(cudaStatus));
        printf("Description: %s\n", hipGetErrorString(cudaStatus));
    }
    for(int i = 0; i < 2*PLAYER_NUM*ANGLE_NUM*DIST_NUM*PLAYER_NUM; ++i)
        result[i] = pointInfor[i];

//    cudaEventRecord(stop);
//    cudaEventSynchronize(stop);

//    float milliseconds = 0;
//    cudaEventElapsedTime(&milliseconds, start, stop);
//    printf("Time: %.5f ms\n", milliseconds);
    hipFree(pointInfor);
}

//extern "C" void PosScore(Player* players, Point* ballPos, Point* bestPositions) {
//    scoreAndPoint *allScore;
//    cudaMallocManaged((void**)&allScore, BLOCK_X_FOR_POS_SCORE * BLOCK_Y_FOR_POS_SCORE * THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE * sizeof(scoreAndPoint));
////    cudaEvent_t start, stop;
////    cudaEventCreate(&start);
////    cudaEventCreate(&stop);
////    cudaEventRecord(start);

//    dim3 blocks(BLOCK_X_FOR_POS_SCORE, BLOCK_Y_FOR_POS_SCORE);
//    dim3 threads(THREAD_X_FOR_POS_SCORE, THREAD_Y_FOR_POS_SCORE);
//    calculateAllPosScore<<< blocks, threads >>> (players, ballPos, allScore);
//    cudaDeviceSynchronize();

//    sortPosScore<<< blocks, threads >>> (allScore);
//    cudaDeviceSynchronize();

//    cudaError_t cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess){
//        printf("CUDA ERROR: %d\n", (int)cudaStatus);
//        printf("Error Name: %s\n", cudaGetErrorName(cudaStatus));
//        printf("Description: %s\n", cudaGetErrorString(cudaStatus));
//    }
////    cudaEventRecord(stop);
////    cudaEventSynchronize(stop);
////    float milliseconds = 0;
////    cudaEventElapsedTime(&milliseconds, start, stop);
////    printf("Time: %.5f ms\n", milliseconds);

//    for(int i = 0; i < BLOCK_X_FOR_POS_SCORE * BLOCK_Y_FOR_POS_SCORE; i++) {
//        bestPositions[i] = allScore[i * THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE].p;
////        printf("(%lf, %lf)\n", allScore[i * THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE].p.x, allScore[i * THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE].p.y);
//    }
//    cudaFree(allScore);
//    return;
//}

// attack threat Evaluation Function for Run Pos
// # attack
// 1.距离对方球门的距离 2.射门有效角度 3.跟球的距离 4.对方车到传球线的距离 5.对方车到接球点的距离
// # defence
// !!!!!!!!!!!!!!!!!!!!!! 可以根據場上形式使用不同的公式
//__device__ float CUDA_evaluateFunc(Point candidate, Point ballPos, Player* enemy, Player receiver)
//{
//    float score = -INITIAL_VALUE;
//    // 1.距离对方球门的距离
//    Point goal;
//    goal.x = 600;
//    goal.y = 0;
//    float distToGoal = sqrt((candidate.x - goal.x) * (candidate.x - goal.x) + (candidate.y - goal.y) * (candidate.y - goal.y));

//    // 2.射门有效角度
//    Point leftGoalPost;
//    Point rightGoalPost;
//    leftGoalPost.x = rightGoalPost.x = 600;
//    leftGoalPost.y = -60;
//    rightGoalPost.y = 60;
//    float leftDir = atan2((candidate.y - leftGoalPost.y) , (candidate.x - leftGoalPost.x));
//    float rightDir = atan2((candidate.y - rightGoalPost.y) , (candidate.x - rightGoalPost.x));
//    float shootAngle = fabs(leftDir - rightDir);
//    shootAngle = shootAngle > PI ? 2*PI - shootAngle : shootAngle;

//    // 3.跟球的距离
//    float distToBall = sqrt((candidate.x - ballPos.x) * (candidate.x - ballPos.x) + (candidate.y - ballPos.y) * (candidate.y - ballPos.y));
//    // 4.对方车到传球线的距离
//    float distToPassLine = INITIAL_VALUE;
//    for (int i=0; i < MAX_PLAYER; i++) {
//        if(enemy[i].isValid){
//            Point projection = projectionPointToLine(candidate, ballPos, enemy[i].Pos);
//            // 判断是否在线段之间
//            if(projection.x > Min(ballPos.x, candidate.x) && projection.x < Max(ballPos.x, candidate.x)){
//                float dist = sqrt((projection.x - enemy[i].Pos.x) * (projection.x - enemy[i].Pos.x) + (projection.y - enemy[i].Pos.y) * (projection.y - enemy[i].Pos.y));
//                if(dist < distToPassLine)
//                    distToPassLine = dist;
//            }
//        }
//    }
//    // 5.对方车到接球点的距离
//    float distToEnemy = INITIAL_VALUE;
//    for (int i=0; i < MAX_PLAYER; i++) {
//        if(enemy[i].isValid){
//            float dist = sqrt((candidate.x - enemy[i].Pos.x) * (candidate.x - enemy[i].Pos.x) + (candidate.y - enemy[i].Pos.y) * (candidate.y - enemy[i].Pos.y));
//            if(dist < distToEnemy)
//                distToEnemy = dist;
//        }
//    }

//    // 当满足最低要求时计算得分
//    if(distToGoal < maxDistToGoal && shootAngle >= minShootAngle && distToBall < maxDistToBall
//            && distToPassLine >= minDistToPassLine && distToEnemy >= minDistToEnemy){
//        // 归一化处理
//        distToGoal = 1 - distToGoal/maxDistToGoal;
//        shootAngle = shootAngle/maxShootAngle;
//        distToBall = 1 - distToBall/maxDistToBall;
//        distToPassLine = distToPassLine/maxDistToPassLine;
//        distToEnemy = distToEnemy/maxDistToEnemy;

//        // 计算得分
//        score = weight1*distToGoal + weight2*shootAngle + weight3*distToBall + weight4*distToPassLine + weight5*distToEnemy;
//    }
//    return score;
//}

//__global__ void calculateAllPosScore(Player* Players, Point* ballPos, scoreAndPoint* allScore) {
//    float blockLength = PITCH_LENGTH / gridDim.x;
//    float blockWidth = PITCH_WIDTH / gridDim.y;
//    float threadLength = blockLength / blockDim.x;
//    float threadWidth = blockWidth / blockDim.y;
//    int blockIndex = gridDim.x * blockIdx.y + blockIdx.x;
//    int threadIndex = blockDim.x * threadIdx.y + threadIdx.x;
//    int allScoreIndex = blockIndex * blockDim.x * blockDim.y + threadIndex;
//    __syncthreads();
//    allScore[allScoreIndex].p.x = blockLength * blockIdx.x + threadLength / 2 + threadLength * threadIdx.x - PITCH_LENGTH / 2;
//    allScore[allScoreIndex].p.y = blockWidth * blockIdx.y + threadWidth / 2 + threadWidth * threadIdx.y - PITCH_WIDTH / 2;
//    __syncthreads();
//    if(IsInPenalty(allScore[allScoreIndex].p))
//        allScore[allScoreIndex].score = INITIAL_VALUE;
//    else
//        allScore[allScoreIndex].score = CUDA_evaluateFunc(allScore[allScoreIndex].p, *ballPos, Players, Players[MAX_PLAYER]);
//    __syncthreads();
//}

//__global__ void sortPosScore(scoreAndPoint *allScore) {
//    __shared__ scoreAndPoint scoreBlock[THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE];
//    int blockIndex = gridDim.x * blockIdx.y + blockIdx.x;
//    int threadIndex = blockDim.x * threadIdx.y + threadIdx.x;
//    int allScoreIndex = blockIndex * blockDim.x * blockDim.y + threadIndex;
//    scoreAndPoint temp;
//    scoreBlock[threadIndex] = allScore[allScoreIndex];
//    __syncthreads();
//    //并行地按照从大到小的次序进行排列
//    bool even = true;
//    for(int i = 0; i < THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE; i++) {
//        if(threadIndex < THREAD_X_FOR_POS_SCORE * THREAD_Y_FOR_POS_SCORE - 1 && even && scoreBlock[threadIndex].score < scoreBlock[threadIndex + 1].score) {
//            temp = scoreBlock[threadIndex + 1];
//            scoreBlock[threadIndex + 1] = scoreBlock[threadIndex];
//            scoreBlock[threadIndex] = temp;
//        }
//        else if(threadIndex > 0 && !even && scoreBlock[threadIndex].score > scoreBlock[threadIndex - 1].score) {
//            temp = scoreBlock[threadIndex];
//            scoreBlock[threadIndex] = scoreBlock[threadIndex - 1];
//            scoreBlock[threadIndex - 1] = temp;
//        }
//        even = !even;
//        __syncthreads();
//    }
//    allScore[allScoreIndex] = scoreBlock[threadIndex];
//    __syncthreads();
//}
